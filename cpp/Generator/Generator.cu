#include <bits/stdc++.h>
#include <iostream>
#include <string>
#include <sstream>
#include <iterator>


#include "Generator.h"


#ifndef GRID_VAL
# define GRID_VAL(arr, width, col, row) ((arr)[((width) * (row)) + (col)])
#endif

// TODO: Get based on GPU
#ifndef BLOCK_SIZE
# define BLOCK_SIZE 1024
#endif

#ifndef NUM_BLOCKS
# define NUM_BLOCKS 100
#endif

std::vector<std::vector<char> > generateCrossword(std::vector<std::string> dictionary)
{
    int width  = 12;
    int height = 8;
    // internally, we'll store the crossword as a 1 dimensional array so we can
    // access this from the GPU more easily
    // the crossword itself is very small, so finding contiguous memory will not
    // be an issue
    char result[width * height];

    // clear out the current array
    for (int i = 0; i < width * height; i++) {
        result[i] = ' ';
    }

    int numCrosswordWords = 1; // TODO: do this dynamically

    std::vector<std::string> candidates = dictionary;

    for (int iteration = 0; iteration < numCrosswordWords; iteration++) {
        // we need to organize the candidate words in GPU memory in a way that each
        // thread knows how to access the words it needs to work on, and can
        // return the index of the best word
        std::vector<std::vector<std::string> > perThreadCandidates;
        for (int i = 0; i < BLOCK_SIZE * NUM_BLOCKS; i++) {
            std::vector<std::string> candidates;
            // use strides of length BLOCK_SIZE * NUM_BLOCKS to determine this
            // thread's words
            for (int j = 0; j < dictionary.size(); j += BLOCK_SIZE * NUM_BLOCKS) {
                candidates.push_back(dictionary[i + j]);
            }
            perThreadCandidates.push_back(candidates);
        }

        int totalChars = 0;
        std::vector<std::string> joinedPerThreadCandidates;
        std::vector<int> threadCandidatesIndices;// index of the string for each candidate
        std::vector<int> joinedPerThreadCandidatesLengths;
        for (int i = 0; i < perThreadCandidates.size(); i++) {
            std::vector<std::string> candidates = perThreadCandidates[i];
            std::ostringstream imploded; // will include all candidates joined
            // by the delimiter with one at the end
            std::copy(candidates.begin(), candidates.end(),
              std::ostream_iterator<std::string>(imploded, "\n"));
            int joinedLength = 0;
            for (int j = 0; j < candidates.size(); j++) {
                joinedLength += candidates[j].length() + 1; // add 1 for delimiter
            }
            threadCandidatesIndices.push_back(totalChars);
            // std::cout << '\n' << totalChars;
            totalChars += joinedLength;
            joinedPerThreadCandidatesLengths.push_back(joinedLength);
            joinedPerThreadCandidates.push_back(imploded.str());
        }

        std::ostringstream allConcatenatedCandidates; // will include all candidates across all threads joined
        std::copy(joinedPerThreadCandidates.begin(), joinedPerThreadCandidates.end(),
          std::ostream_iterator<std::string>(allConcatenatedCandidates, ""));

        // std::cout << allConcatenatedCandidates.str();

        char * device_crossword; // the copy of the crossword that lives in GPU RAM

        hipMalloc(&device_crossword, (width * height) * sizeof(char));

        hipFree(device_crossword);
    }


    // unpack the result into vectors
    std::vector<std::vector<char> > resultVec;
    for (int y = 0; y < height; y++) {
        std::vector<char> row;
        for (int x = 0; x < width; x++) {
            // GRID_VAL(result, width, x, y) = 65 + x + y;
            row.push_back(GRID_VAL(result, width, x, y));
        }
        resultVec.push_back(row);
    }

    return resultVec;
} // generateCrossword
