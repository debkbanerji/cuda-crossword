#include <bits/stdc++.h>
#include <iostream>
#include <string>
#include <sstream>
#include <iterator>


#include "Generator.h"


#ifndef GRID_VAL
# define GRID_VAL(arr, width, col, row) ((arr)[((width) * (row)) + (col)])
#endif

// TODO: Get based on GPU
#ifndef BLOCK_SIZE
# define BLOCK_SIZE 1024
#endif

#ifndef NUM_BLOCKS
# define NUM_BLOCKS 100
#endif

std::vector<std::vector<char> > generateCrossword(std::vector<std::string> dictionary)
{
    int width  = 12;
    int height = 8;
    // internally, we'll store the crossword as a 1 dimensional array so we can
    // access this from the GPU more easily
    // the crossword itself is very small, so finding contiguous memory will not
    // be an issue
    char result[width * height];

    // clear out the current array
    for (int i = 0; i < width * height; i++) {
        result[i] = ' ';
    }

    int numCrosswordWords = 1; // TODO: do this dynamically

    std::vector<std::string> candidates = dictionary;

    for (int iteration = 0; iteration < numCrosswordWords; iteration++) {
        // we need to organize the candidate words in GPU memory in a way that each
        // thread knows how to access the words it needs to work on, and can
        // return the index of the best word
        std::vector<std::vector<std::string> > perThreadCandidates;
        for (int i = 0; i < BLOCK_SIZE * NUM_BLOCKS; i++) {
            std::vector<std::string> candidates;
            // use strides of length BLOCK_SIZE * NUM_BLOCKS to determine this
            // thread's words
            for (int j = 0; j < dictionary.size(); j += BLOCK_SIZE * NUM_BLOCKS) {
                candidates.push_back(dictionary[i + j]);
            }
            perThreadCandidates.push_back(candidates);
        }

        int totalChars = 0;
        std::vector<std::string> joinedPerThreadCandidates;
        std::vector<int> threadCandidatesIndices;// index of the string for each candidate
        std::vector<int> joinedPerThreadCandidatesLengths;
        for (int i = 0; i < perThreadCandidates.size(); i++) {
            std::vector<std::string> candidates = perThreadCandidates[i];
            std::ostringstream imploded; // will include all candidates joined
            // by the delimiter with one at the end
            std::copy(candidates.begin(), candidates.end(),
              std::ostream_iterator<std::string>(imploded, "\n"));
            int joinedLength = 0;
            for (int j = 0; j < candidates.size(); j++) {
                joinedLength += candidates[j].length() + 1; // add 1 for delimiter
            }
            threadCandidatesIndices.push_back(totalChars);
            // std::cout << '\n' << totalChars;
            totalChars += joinedLength;
            joinedPerThreadCandidatesLengths.push_back(joinedLength);
            joinedPerThreadCandidates.push_back(imploded.str());
        }

        std::ostringstream allConcatenatedCandidates; // will include all candidates across all threads joined
        std::copy(joinedPerThreadCandidates.begin(), joinedPerThreadCandidates.end(),
          std::ostream_iterator<std::string>(allConcatenatedCandidates, ""));

        std::string allConcatenatedCandidatesStr = allConcatenatedCandidates.str();
        char allConcatenatedCandidatesArray[allConcatenatedCandidatesStr.length()];
        strcpy(allConcatenatedCandidatesArray, allConcatenatedCandidatesStr.c_str());

        int threadCandidatesIndicesArray[threadCandidatesIndices.size()];
        std::copy(threadCandidatesIndices.begin(), threadCandidatesIndices.end(), threadCandidatesIndicesArray);

        // std::cout << allConcatenatedCandidates.str();

        char * deviceCrossword;
        char * deviceAllConcatenatedCandidatesArray;
        int * deviceThreadCandidatesIndicesArray;

        hipMalloc(&deviceCrossword, (width * height) * sizeof(char));
        hipMalloc(&deviceAllConcatenatedCandidatesArray, allConcatenatedCandidatesStr.length() * sizeof(char));
        hipMalloc(&deviceThreadCandidatesIndicesArray, threadCandidatesIndices.size() * sizeof(int));

        // TODO: kernel work
        // TODO: Allocate array for results

        hipFree(deviceCrossword);
        hipFree(deviceAllConcatenatedCandidatesArray);
        hipFree(deviceThreadCandidatesIndicesArray);

        // TODO: Update result based on GPU work
    }


    // unpack the result into vectors
    std::vector<std::vector<char> > resultVec;
    for (int y = 0; y < height; y++) {
        std::vector<char> row;
        for (int x = 0; x < width; x++) {
            // GRID_VAL(result, width, x, y) = 65 + x + y;
            row.push_back(GRID_VAL(result, width, x, y));
        }
        resultVec.push_back(row);
    }

    return resultVec;
} // generateCrossword
